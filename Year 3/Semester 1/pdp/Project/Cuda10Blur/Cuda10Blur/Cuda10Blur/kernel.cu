#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <string>
#include "process_image.h"


__device__
int
AccessPixel(
	_In_ uchar* Arr,
	_In_ int Col,
	_In_ int Row,
	_In_ int K,
	_In_ int Width,
	_In_ int Height
)
{

	int kernel[3][3] = { 1, 2, 1,
				   2, 4, 2,
				   1, 2, 1 };

	int sum = 0;
	int sumKernel = 0;

	for (int j = -1; j <= 1; j++)
	{
		for (int i = -1; i <= 1; i++)
		{
			if ((Row + j) >= 0 && (Row + j) < Height && (Col + i) >= 0 && (Col + i) < Width)
			{
				int color = Arr[(Row + j) * 3 * Width + (Col + i) * 3 + K];
				sum += color * kernel[i + 1][j + 1];
				sumKernel += kernel[i + 1][j + 1];
			}
		}
	}

	return sum / sumKernel;
}

__global__
void CudaPixelWorker(
	_In_ unsigned char* Img,
	_Out_ unsigned char* Res,
	_In_ int Width,
	_In_ int Height
)
{
	// blockDim.x - dimension of a block (256; must me multiple of 32)
	// blockIdx.x - current block index
	// threadIdx.x - current id of the thread in the 1D array (nvidia GRID)

	// gridDim.x - total blocks

	int startIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int row = startIndex; row < Height; row += stride)
	{
		for (int col = 0; col < Width; col++)
		{
			for (int k = 0; k < 3; k++)
			{
				Res[3 * row * Width + 3 * col + k] = AccessPixel(Img, col, row, k, Width, Height);
			}
		}
	}
}

void
GaussianBlur2D(
	_In_ const std::string& FileName
)
{
	cv::Mat3b img = cv::imread(FileName, cv::IMREAD_COLOR);
	cv::Mat3b out(img.rows, img.cols);

	uchar *buffer;
	uchar *buffer2;

	if (!img.data) {
		printf("Failed to read image\n");
		exit(2);
	}
	hipError_t hipError_t;
	hipError_t = hipMallocManaged(&buffer, img.rows * img.cols * 3 * sizeof(uchar));
	hipError_t = hipMallocManaged(&buffer2, img.rows * img.cols * 3 * sizeof(uchar));

	if (hipError_t != hipSuccess)
	{
		std::cout << "[Error] - Cuda Error!" << std::endl;
	}

	out.data = buffer2;

	hipMemcpy(buffer, img.data, img.rows * img.cols * 3, hipMemcpyHostToDevice);
	img.data = buffer;

	int threadsPerBlock = 256;
	int totalBlocks = (img.rows + threadsPerBlock - 1) / threadsPerBlock;
	CudaPixelWorker<<<totalBlocks, threadsPerBlock>>>(img.data, out.data, img.cols, img.rows);
	hipDeviceSynchronize();

	std::cout << "Trying to save image to output file" << std::endl;
	cv::imwrite("output.bmp", out);

	hipFree(buffer);
	hipFree(buffer2);
}

int main(int argc, char** argv)
{
	std::string fileName;

	std::cout << "Cuda started" << std::endl;
	Usage(argc, argv, fileName);
	std::cout << "Filename: " << fileName << std::endl;

	GaussianBlur2D(fileName);

	return 0;
}