#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <math.h>
#include <string>

#include "process_image.h"


// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	printf("%d\n", index);
	//std::cout << std::endl;

	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}


__device__
int
accessPixel(
	unsigned char * arr,
	int col,
	int row,
	int k,
	int width,
	int height
)
{

	int kernel[3][3] = { 1, 2, 1,
				   2, 4, 2,
				   1, 2, 1 };

	int sum = 0;
	int sumKernel = 0;

	for (int j = -1; j <= 1; j++)
	{
		for (int i = -1; i <= 1; i++)
		{
			if ((row + j) >= 0 && (row + j) < height && (col + i) >= 0 && (col + i) < width)
			{
				int color = arr[(row + j) * 3 * width + (col + i) * 3 + k];
				//printf("%d\n", color);
				sum += color * kernel[i + 1][j + 1];
				sumKernel += kernel[i + 1][j + 1];
			}
		}
	}

	return sum / sumKernel;
}

__global__
void CudaPixelWorker(
	_In_ unsigned char* Img,
	_Out_ unsigned char* Res,
	_In_ int Width,
	_In_ int Height
)
{
	// blockDim.x - dimension of a block (256; must me multiple of 32)
	// blockIdx.x - current block index
	// threadIdx.x - current id of the thread in the 1D array (nvidia GRID)

	// gridDim.x - total blocks

	int startIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;


	for (int row = startIndex; row < Height; row += stride)
	{
		for (int col = 0; col < Width; col++)
		{
			for (int k = 0; k < 3; k++)
			{
				Res[3 * row * Width + 3 * col + k] = accessPixel(Img, col, row, k, Width, Height);
			}
		}
	}
}

void
GaussianBlur2D(
	_In_ const std::string& FileName
)
{
	cv::Mat3b img = cv::imread(FileName, cv::IMREAD_COLOR);
	cv::Mat3b out(img.rows, img.cols);

	uchar *buffer;
	uchar *buffer2;

	if (!img.data) {
		printf("Failed to read image\n");
		exit(2);
	}
	hipError_t hipError_t;
	hipError_t = hipMallocManaged(&buffer, img.rows * img.cols * 3 * sizeof(uchar));
	hipError_t = hipMallocManaged(&buffer2, img.rows * img.cols * 3 * sizeof(uchar));

	if (hipError_t != hipSuccess)
	{
		std::cout << "[Error] - Cuda Error!" << std::endl;
	}

	out.data = buffer2;
	hipMemcpy(buffer, img.data, img.rows * img.cols * 3, hipMemcpyHostToDevice);
	img.data = buffer;

	int threadsPerBlock = 256;
	int totalBlocks = (img.rows + threadsPerBlock - 1) / threadsPerBlock;
	CudaPixelWorker<<<totalBlocks, threadsPerBlock>>>(img.data, out.data, img.cols, img.rows);
	hipDeviceSynchronize();
	std::cout << "Copying back out.data to img.data" << std::endl;

	//memcpy(img.data, out.data, img.rows * img.cols * 3);

	std::cout << "Trying to save image to output file" << std::endl;
	cv::imwrite("output.bmp", out);

	hipFree(buffer);
	hipFree(buffer2);
}


int main(int argc, char** argv)
{
	std::string fileName;

	std::cout << "Cuda started" << std::endl;
	Usage(argc, argv, fileName);
	std::cout << "Filename: " << fileName << std::endl;

	GaussianBlur2D(fileName);

	//int N = 1 << 20;
	//float *x, *y;

	//// Allocate Unified Memory � accessible from CPU or GPU
	//hipMallocManaged(&x, N * sizeof(float));
	//hipMallocManaged(&y, N * sizeof(float));

	//// initialize x and y arrays on the host
	//for (int i = 0; i < N; i++) {
	//	x[i] = 1.0f;
	//	y[i] = 2.0f;
	//}

	//// Run kernel on 1M elements on the GPU

	//int noThreadsPerBlock = 256;
	//int totalBlocks = (N + noThreadsPerBlock - 1) / noThreadsPerBlock;

	//add<<<totalBlocks, noThreadsPerBlock>>> (N, x, y);

	//// Wait for GPU to finish before accessing on host
	//hipDeviceSynchronize();

	//// Check for errors (all values should be 3.0f)
	//float maxError = 0.0f;
	//for (int i = 0; i < N; i++)
	//	maxError = fmax(maxError, fabs(y[i] - 3.0f));
	//std::cout << "Max error: " << maxError << std::endl;

	//// Free memory
	//hipFree(x);
	//hipFree(y);

	return 0;
}