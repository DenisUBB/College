#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

// Must be a power of 2
#define POLYNOM_SIZE 4

#define FINAL_POLYNOM_SIZE (POLYNOM_SIZE * 2 - 1)

typedef unsigned long long int  uint64_t;

static
void
PrintPoly(
	_In_ const uint64_t* Pol
)
{
	size_t n = POLYNOM_SIZE * 2 - 1;

	for (size_t i = 0; i < n; i++)
	{
		if (Pol[i] == 0) continue;

		std::cout << Pol[i];
		if (i != 0)
			std::cout << "x^" << i;

		if (i != n - 1 && Pol[i + 1] != 0)
			std::cout << " + ";
	}

	std::cout << std::endl;
}

void
GenerateRandomNums(uint64_t* a, uint64_t* b)
{
	a[0] = 5;
	a[1] = 0;
	a[2] = 10;
	a[3] = 6;

	b[0] = 1;
	b[1] = 2;
	b[2] = 4;
	b[3] = 3;
}

__global__
void
CudaNaiveMultiplication(
	_In_ uint64_t* A,
	_In_ uint64_t* B,
	_In_ uint64_t* Prod,
	_In_ int End)
{
	int coefficientToStart = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = coefficientToStart; i < End; i += stride)
	{
		// eg for coefficient 3, we would have (a[0] * b[3]), (a[1] * b[2]), (a[2] * B[1]), (a[3] * b[0]))

		for (int coeffInA = 0; coeffInA <= i; ++coeffInA)
		{
			int coeffInB = i - coeffInA;
			Prod[i] += (A[coeffInA] * B[coeffInB]);
		}
	}
}

void
CudaPolynomMultiplication(
	_In_ uint64_t* A,
	_In_ uint64_t* B,
	_In_ uint64_t* Prod
	)
{
	printf("Cuda naive algorithm polynom multiplication\n");

	int noThreadsPerBlock = 32;
	int totalBlocks = ((POLYNOM_SIZE * 2 - 1) + noThreadsPerBlock - 1) / noThreadsPerBlock;
	CudaNaiveMultiplication <<<1, 32>>> (A, B, Prod, POLYNOM_SIZE * 2 - 1);
	hipDeviceSynchronize();

	PrintPoly(Prod);
}

void AllocateBuffers(
	_In_ uint64_t* &A,
	_In_ uint64_t* &B,
	_In_ uint64_t* &Prod
	)
{
	int polSize;
	hipError_t hipError_t;

	polSize = POLYNOM_SIZE * 2 - 1;
	hipError_t = hipMallocManaged(&A, polSize * sizeof(uint64_t));
	if (hipError_t != hipSuccess)
	{
		printf("[ERROR] Couldn not allocate buffer. Aborting\n");
		return;
	}
	hipMemset(A, 0, polSize * sizeof(uint64_t));

	hipError_t = hipMallocManaged(&B, polSize * sizeof(uint64_t));
	if (hipError_t != hipSuccess)
	{
		printf("[ERROR] Couldn not allocate buffer. Aborting\n");
		hipFree(A);
		return;
	}
	hipMemset(B, 0, polSize * sizeof(uint64_t));

	hipError_t = hipMallocManaged(&Prod, polSize * sizeof(uint64_t));
	if (hipError_t != hipSuccess)
	{
		printf("[ERROR] Couldn not allocate buffer. Aborting\n");
		hipFree(A);
		hipFree(B);
		return;
	}
	hipMemset(Prod, 0, polSize * sizeof(uint64_t));
}

void
FreeBuffers(
	_In_ uint64_t* &A,
	_In_ uint64_t* &B,
	_In_ uint64_t* &Prod
	)
{
	hipFree(A);
	hipFree(B);
	hipFree(Prod);
}


__device__
void
_Karatsuba(
	_In_ const vector<int> &A,
	_In_ const vector<int> &B,
	_Out_ vector<int> &C
)
{
	if (A.size() == 1 && B.size() == 1)
	{
		C[0] = A[0] * B[0];
		return;
	}

	size_t half = A.size() / 2;

	vector<int> A0(A.begin(), A.begin() + half);    // A0
	vector<int> A1(A.begin() + half, A.end());      // A1

	vector<int> B0(B.begin(), B.begin() + half);    // B0
	vector<int> B1(B.begin() + half, B.end());      // B1

	vector<int> C1(A0.size() + B0.size() - 1);      // C1
	vector<int> C2(A1.size() + B1.size() - 1);      // C2

	_Karatsuba(A0, B0, C1);     // C1 = A0 * B0 - by recursive call
	_Karatsuba(A1, B1, C2);     // C2 = A1 * B1 - by recursive call

	for (size_t i = 0; i < C1.size(); ++i)
	{
		C[i] += C1[i];                        // C = C1;
	}

	for (size_t i = 0; i < A1.size(); ++i)
	{
		A0[i] += A1[i];                       // C3 = A0 + A1
		B0[i] += B1[i];                       // C4 = B0 + B1
	}
	vector<int> C5(A0.size() + B0.size() - 1);
	_Karatsuba(A0, B0, C5);     // C5 = C3 * C4 - by recursive call

	for (size_t i = 0; i < C5.size(); ++i)          // At this point C = C1;
	{
		C[i + half] += C5[i] - C1[i] - C2[i];     // C += C6; => C = C1 + C6; where C6 = C5 - C1 - C2
	}

	for (size_t i = 0; i < C2.size(); ++i)          // At this point C = C1 + C6*X^(n/2)
	{
		C[i + 2 * half] += C2[i];                   // C = C1 + C6*X^(n/2) + C2*X^n
	}
}

__global__
void
CudaKaratsuba(
	_In_ uint64_t* A,
	_In_ uint64_t* B,
	_In_ uint64_t* Prod
	)
{
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = threadId; i < FINAL_POLYNOM_SIZE; i += stride)
	{
		unsigned char* temp;
		hipMallocManaged(&temp, FINAL_POLYNOM_SIZE * sizeof(uint64_t));
		hipMemset(temp, 0, FINAL_POLYNOM_SIZE * sizeof(uint64_t));
		temp[i] = A[i];


	}
}

void
CudaPolynomMultiplicationKaratsuba(
	_In_ uint64_t* &A,
	_In_ uint64_t* &B,
	_In_ uint64_t* &Prod
	)
{
}


int main(int argc, char** argv)
{
	uint64_t *a, *b, *result;
	
	AllocateBuffers(a, b, result);
	GenerateRandomNums(a, b);

	CudaPolynomMultiplication(a, b, result);

	hipMemset(result, 0, FINAL_POLYNOM_SIZE * sizeof(uint64_t));

	CudaPolynomMultiplicationKaratsuba(a, b, result);

	FreeBuffers(a, b, result);
	return 0;
}